#include "hip/hip_runtime.h"
﻿#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <string>
#include <iostream>
#include <fstream>
#include <nlohmann/json.hpp>
#include <thrust/device_vector.h>
#include <thrust/reduce.h>

using json = nlohmann::json;
using namespace std;

// myFunction<<<block_count, thread_count>>>();

// __global__ - functions that can be called from CPU (host) and be executed on GPU (device)
// __device__ - functions that can be called and executed from GPU (Device)

// Constants FOR CPU
const int INPUT_FILE_DATA_COUNT = 300;
const int WORD_LENGTH = 25;

// Constats FOR GPU
__device__ const int GPU_WORD_LENGTH = 25;

class Anime{
public:
    char Name[WORD_LENGTH];
    int ReleaseYear;
    double Score;
};

__device__ int HashFunction(int length) {
    int hash = length; // Start with the input value
    hash = (hash * 31) ^ (hash >> 3); // Multiply and XOR with a shifted value
    hash = (hash * 17) + 12345;       // Mix it further with another constant
    hash = hash % 5000;         // Ensure it fits into [0, 4999]
    printf("%d\n", hash);
    return hash;
}

__global__ void CudaCalculations(const Anime* anime, char* resultCharArray, const int* inputDataLength, int* resultIndexCount) {
    int arrayIndex = blockIdx.x * blockDim.x + threadIdx.x;

    //Check if operation is outside of array
    if (arrayIndex >= *inputDataLength)
        return;

    Anime element = anime[arrayIndex];
    int hash = HashFunction(element.Score);
    

    //Filter check, if fail = not added
    if (hash >= 2500)
        return;

    int index = atomicAdd(resultIndexCount, 1); //Adds a word
    index = index * GPU_WORD_LENGTH; //Recalculate the next word start

    for (int charIndex = 0; element.Name[charIndex] != 0; charIndex++, index++) {
        resultCharArray[index] = element.Name[charIndex];
    }

    //Add a delimiter
    resultCharArray[index] = '-';
    index++;

    //Add last range
    if (hash >= 1250) {
        resultCharArray[index++] = '1';
        resultCharArray[index++] = '2';
        resultCharArray[index++] = '5';
        resultCharArray[index++] = '0';
        resultCharArray[index++] = '>';
        resultCharArray[index++] = '=';
    }
    else {
        resultCharArray[index++] = '1';
        resultCharArray[index++] = '2';
        resultCharArray[index++] = '5';
        resultCharArray[index++] = '0';
        resultCharArray[index++] = '<';
    }
}

int main() {
    printf("Program started\n");

    string inputFileName = "Algimantas_Vezevicius_data_3.json";
    string outputFileName = "Algimantas_Vezevicius_rez_3.txt";

    //reading
    ifstream inputFile(inputFileName);
    json jsonData;
    inputFile >> jsonData;

    const int inputDataCount = jsonData["AnimeData"].size();
    cout << "Read Data Size: " << inputDataCount << endl;

    Anime anime[INPUT_FILE_DATA_COUNT];
    for (int i = 0; i < jsonData["AnimeData"].size(); i++) {
        json currentJson = jsonData["AnimeData"][i];

        string name = currentJson["Name"];
        int year = currentJson["ReleaseYear"];
        double rating = currentJson["Rating"];

        anime[i].ReleaseYear = year;
        anime[i].Score = rating;
        strcpy(anime[i].Name, name.c_str());
    }
    printf("Data reading finished\n");

    //ponters for memory transfer for CUDA to VRAM - CUDA
    int resultCounter = 0;
    char* gpu_result;
    int* gpu_resultCounter;
    int* gpu_inputSizeCount;
    Anime* gpu_inputData;

    //Memory allocation to GPU VRAM
    hipMalloc(&gpu_inputData, sizeof(anime));
    hipMalloc(&gpu_result, sizeof(char) * WORD_LENGTH * INPUT_FILE_DATA_COUNT);
    hipMalloc(&gpu_resultCounter, sizeof(int));
    hipMalloc(&gpu_inputSizeCount, sizeof(int));

    //CPU RAM to GPU VRAM copy allocation
    hipMemcpy(gpu_inputData, anime, sizeof(anime), hipMemcpyHostToDevice);
    hipMemcpy(gpu_resultCounter, &resultCounter, sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(gpu_inputSizeCount, &INPUT_FILE_DATA_COUNT, sizeof(int), hipMemcpyHostToDevice);

    //Optimal block and thread size calculation --- CUDA
    int thread_count = 32 * 2;
    int block_count = (inputDataCount / thread_count) + 1;
    int outputDataCount = thread_count * block_count;
    cout << "Thread Count: " << thread_count << endl;
    cout << "Block Count: " << block_count << endl;
    cout << "End Array Size: " << outputDataCount << endl;

    //GPU work --- CUDA
    CudaCalculations <<<block_count, thread_count>>> (gpu_inputData, gpu_result, gpu_inputSizeCount, gpu_resultCounter);
    hipDeviceSynchronize();

    //Copying GPU VRAM to CPU RAM --- CUDA
    char results[WORD_LENGTH * INPUT_FILE_DATA_COUNT];
    hipMemcpy(results, gpu_result, sizeof(results), hipMemcpyDeviceToHost);
    hipMemcpy(&resultCounter, gpu_resultCounter, sizeof(int), hipMemcpyDeviceToHost);

    //Memory release so to not cause a leak --- CUDA
    hipFree(gpu_result);
    hipFree(gpu_resultCounter);
    hipFree(gpu_inputData);

    //Results
    cout << "Result count: " << resultCounter << endl;
    ofstream outputFile(outputFileName);
    for (int i = 0; i < WORD_LENGTH * resultCounter; i++) {
        if ((i + 1) % WORD_LENGTH == 0) //end of word
            outputFile << endl;
        else
            outputFile << results[i];
    }

    outputFile.close();

    printf("Program finished\n");
    return 0;
}
